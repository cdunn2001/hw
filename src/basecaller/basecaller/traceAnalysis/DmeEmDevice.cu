#include "hip/hip_runtime.h"
// Copyright (c) 2021, Pacific Biosciences of California, Inc.
//
// All rights reserved.
//
// THIS SOFTWARE CONSTITUTES AND EMBODIES PACIFIC BIOSCIENCES' CONFIDENTIAL
// AND PROPRIETARY INFORMATION.
//
// Disclosure, redistribution and use of this software is subject to the
// terms and conditions of the applicable written agreement(s) between you
// and Pacific Biosciences, where "you" refers to you or your company or
// organization, as applicable.  Any other disclosure, redistribution or
// use is prohibited.
//
// THIS SOFTWARE IS PROVIDED BY PACIFIC BIOSCIENCES AND ITS CONTRIBUTORS "AS
// IS" AND ANY EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO,
// THE IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR
// PURPOSE ARE DISCLAIMED. IN NO EVENT SHALL PACIFIC BIOSCIENCES OR ITS
// CONTRIBUTORS BE LIABLE FOR ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL,
// EXEMPLARY, OR CONSEQUENTIAL DAMAGES (INCLUDING, BUT NOT LIMITED TO,
// PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES; LOSS OF USE, DATA, OR PROFITS;
// OR BUSINESS INTERRUPTION) HOWEVER CAUSED AND ON ANY THEORY OF LIABILITY,
// WHETHER IN CONTRACT, STRICT LIABILITY, OR TORT (INCLUDING NEGLIGENCE OR
// OTHERWISE) ARISING IN ANY WAY OUT OF THE USE OF THIS SOFTWARE, EVEN IF
// ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
//
//  Description:
//  Defines members of class DmeEmDevice.

#include "DmeEmDevice.h"

#include <algorithm>
#include <limits>

#include <common/IntInterval.h>

#include <common/cuda/memory/AllocationViews.cuh>
#include <common/cuda/PBCudaSimd.cuh>
#include <common/cuda/streams/LaunchManager.cuh>

#include <dataTypes/configs/BasecallerDmeConfig.h>
#include <dataTypes/configs/AnalysisConfig.h>
#include <basecaller/traceAnalysis/DmeDiagnostics.h>

///////////////////////////////////////////////////////////////////
// TODO There are a lot of commented out PBAssert statements below,
//      as they will not work on the GPU.  PTSD-267 will hopefully
//      result in a replacement that can be slotted in
///////////////////////////////////////////////////////////////////

using std::numeric_limits;
using namespace PacBio::Cuda;
using namespace PacBio::Cuda::Utility;
using namespace PacBio::Mongo::Data;

using FrameIntervalType = PacBio::Mongo::IntInterval<FrameIndexType>;


namespace PacBio {
namespace Mongo {
namespace Basecaller {

namespace {

struct AnalogMode
{
    char baseLabel;
    float relAmplitude;
    float excessNoiseCV;
    float interPulseDistance;   // seconds
    float pulseWidth;           // seconds
    float pw2SlowStepRatio;
    float ipd2SlowStepRatio;
};

__constant__ CoreDMEstimator::Configuration baseConfig;

// Wrapping all the static configurations into a single struct,
// as that will be easier to upload to the GPU.
struct StaticConfig
{
    CudaArray<AnalogMode, 4> analogs;
    float analogMixFracThresh0_;
    float analogMixFracThresh1_;
    float confidHalfLife0_;
    float confidHalfLife1_;
    float scaleSnrConfTol_;
    unsigned short emIterLimit_;
    float gTestFactor_;
    bool iterToLimit_;
    float pulseAmpRegCoeff_;
    float snrDropThresh_;
    float snrThresh0_;
    float snrThresh1_;
    float successConfThresh_;
    uint32_t updateMethod_;
    float refSnr_;       // Expected SNR for analog with relative amplitude of 1
};

__constant__ StaticConfig staticConfig;

/// Saturated linear activation function.
/// A fuzzy threshold that ramps from 0 at a to 1 at b.
/// \returns (x - a)/(b - a) clamped to [0, 1] range.
// TODO fix duplication?
__device__ float satlin(float a, float b, float x)
{
    const auto r = (x - a) / (b - a);
    return min(max(r, 0.f), 1.f);
}

using LaneDetModel = Data::LaneModelParameters<PBHalf2, laneSize/2>;

__device__ const AnalogMode& Analog(int i)
{
    return staticConfig.analogs[i];
}

static constexpr auto numBins = DmeEmDevice::LaneHist::numBins;

template <typename VF>
__device__ VF ModelSignalCovar(float excessNoiseCV2, VF sigMean, VF blVar)
{
    blVar += baseConfig.shotVarCoeff * sigMean;

    // Typically, excessNoiseCV2 << 1 and sigMean >> 1. Order multiplications
    // this way to avoid overflow, especially when VF is half-precision.
    blVar += excessNoiseCV2 * sigMean * sigMean;
    return blVar;
}

template <typename VF>
__device__ VF XsnCoeffCVSq(const VF& sigMean, const VF& sigCovar, const VF& blVar)
{
    VF r (sigCovar - blVar);
    r -= sigMean * baseConfig.shotVarCoeff;
    r /= sigMean * sigMean;
    return r;
}

__device__ void UpdateMode(const ZmwAnalogMode& from,
                             ZmwAnalogMode* to,
                             float fraction)
{
    const float a = fraction;
    const float b = 1 - fraction;
    to->mean = a * from.mean + b * to->mean;
    to->var  = a * from.var  + b * to->var;
}

__device__ void UpdateModel0(const ZmwDetectionModel& from,
                              ZmwDetectionModel *to,
                              float fraction)
{
    UpdateMode(from.baseline, &to->baseline, fraction);
    for (int i = 0; i < to->numAnalogs; ++i)
    {
        UpdateMode(from.analogs[i], &to->analogs[i], fraction);
    }
}

__device__ void UpdateModel1(const ZmwDetectionModel& from,
                              ZmwDetectionModel *to,
                              float fraction)
{

    const float a = 1.0f - fraction;
    const float b = fraction;

    auto &tbm = to->baseline;
    auto &obm = from.baseline;

    auto bw = a * tbm.weight + b * obm.weight;
    tbm.weight = bw;

    auto bm = a * tbm.mean + b * obm.mean;
    tbm.mean = bm;

    auto bv = powf(tbm.var, a) * powf(obm.var, b);
    tbm.var = bv;

    // Four analogs with remaining weight equally partitioned 
    auto aw = 0.25f * (1.0f - bw);

    for (int i = 0; i < to->numAnalogs; ++i)
    {
        auto& tdmi = to->analogs[i];
        auto& obmi = from.analogs[i];

        tdmi.weight = aw;

        auto am = powf(tdmi.mean, a) * powf(obmi.mean, b);
        tdmi.mean = am;

        auto cv = Analog(i).excessNoiseCV;
        auto av = ModelSignalCovar(cv*cv, am, bv);
        tdmi.var = av;
    }

}

__device__ void UpdateModel2(const ZmwDetectionModel& from,
                              ZmwDetectionModel *to,
                              float fraction)
{

    const float a = 1.0f - fraction;
    const float b = fraction;

    auto &tbm = to->baseline;
    auto &obm = from.baseline;

    const auto prevBlCovar = tbm.var;

    auto bw = a * tbm.weight + b * obm.weight;
    tbm.weight = bw;

    auto bm = a * tbm.mean + b * obm.mean;
    tbm.mean = bm;

    auto bv = powf(tbm.var, a) * powf(obm.var, b);
    tbm.var = bv;

    // Four analogs with remaining weight equally partitioned 
    auto aw = 0.25f * (1.0f - bw);

    for (int i = 0; i < to->numAnalogs; ++i)
    {
        auto& tdmi = to->analogs[i];
        auto& obmi = from.analogs[i];

        tdmi.weight = aw;

        const auto tXsnCVSq = XsnCoeffCVSq(tdmi.mean, tdmi.var, prevBlCovar);
        const auto oXsnCVSq = XsnCoeffCVSq(obmi.mean, obmi.var, obm.var);
        const auto newXsnCVSq = a * tXsnCVSq  + b * oXsnCVSq;

        auto am = powf(tdmi.mean, a) * powf(obmi.mean, b);
        tdmi.mean = am;

        auto av = ModelSignalCovar(newXsnCVSq, am, bv);
        tdmi.var = av;
    }

}

__device__ void UpdateModel(const ZmwDetectionModel& from,
                              ZmwDetectionModel *to)
{
    float toConfidence = 0;
    assert (from.confidence >= 0.0f);
    assert (toConfidence >= 0.0f);

    const auto confSum = from.confidence + toConfidence;
    const float fraction = confSum > 0.0f ? from.confidence / confSum: 0.f;

    assert (fraction >= 0.0f);
    assert (fraction <= 1.0f);
    //assert ((fraction > 0) | (confSum == Confidence())));
    switch (staticConfig.updateMethod_)
    {
        case 0: UpdateModel0(from, to, fraction); break;
        case 1: UpdateModel1(from, to, fraction); break;
        case 2: UpdateModel2(from, to, fraction); break;
        // hmm, can't throw an exception in device code, so defaulting to 0
        // and doing this explicitly just to prevent confusion
        default: UpdateModel0(from, to, fraction); break;
    }

    to->confidence=confSum;
}

}

DmeEmDevice::DmeEmDevice(uint32_t poolId, unsigned int poolSize)
    : CoreDMEstimator(poolId, poolSize)
{ }

// static
void DmeEmDevice::Configure(const Data::BasecallerDmeConfig &dmeConfig,
                            const Data::AnalysisConfig &analysisConfig)
{
    CoreDMEstimator::Configure(analysisConfig);
    Cuda::CudaCopyToSymbol(&baseConfig, &CoreDMEstimator::Config());

    // TODO: Validate values.
    // TODO: Log settings.
    StaticConfig config;
    auto& movieInfo = analysisConfig.movieInfo;
    for (size_t i = 0; i < movieInfo.analogs.size(); i++)
    {
        config.analogs[i].baseLabel = movieInfo.analogs[i].baseLabel;
        config.analogs[i].relAmplitude = movieInfo.analogs[i].relAmplitude;
        config.analogs[i].excessNoiseCV = movieInfo.analogs[i].excessNoiseCV;
        config.analogs[i].interPulseDistance = movieInfo.analogs[i].interPulseDistance;
        config.analogs[i].pulseWidth = movieInfo.analogs[i].pulseWidth;
        config.analogs[i].pw2SlowStepRatio = movieInfo.analogs[i].pw2SlowStepRatio;
        config.analogs[i].ipd2SlowStepRatio = movieInfo.analogs[i].ipd2SlowStepRatio;
    }

    config.analogMixFracThresh0_ = dmeConfig.AnalogMixFractionThreshold[0];
    config.analogMixFracThresh1_ = dmeConfig.AnalogMixFractionThreshold[1];
    config.confidHalfLife0_ = dmeConfig.ConfidenceHalfLife[0];
    config.confidHalfLife1_ = dmeConfig.ConfidenceHalfLife[1];
    config.scaleSnrConfTol_ = dmeConfig.ScaleSnrConfTol;

    config.emIterLimit_ = dmeConfig.EmIterationLimit;
    config.gTestFactor_ = dmeConfig.GTestStatFactor;
    config.iterToLimit_ = dmeConfig.IterateToLimit;
    config.pulseAmpRegCoeff_  = dmeConfig.PulseAmpRegularization;
    config.snrDropThresh_ = dmeConfig.SnrDropThresh;
    config.snrThresh0_ = dmeConfig.MinAnalogSnrThresh0;
    config.snrThresh1_ = dmeConfig.MinAnalogSnrThresh1;
    config.successConfThresh_ = dmeConfig.SuccessConfidenceThresh;
    config.updateMethod_   = dmeConfig.ModelUpdateMethod;
    config.refSnr_ = movieInfo.refSnr;

    Cuda::CudaCopyToSymbol(&staticConfig, &config);
}

__device__ int TotalCount(const DmeEmDevice::LaneHist& hist)
{
    int count = hist.outlierCountHigh[threadIdx.x] + hist.outlierCountLow[threadIdx.x];
    for (int i = 0; i < numBins; ++i)
    {
        count += hist.binCount[i][threadIdx.x];
    }
    return count;
}

__device__ float Fractile(const DmeEmDevice::LaneHist& hist, float frac)
{
    assert(frac >= 0.f);
    assert(frac <= 1.0f);

    static constexpr auto inf = std::numeric_limits<float>::infinity();

    int totalCount = hist.outlierCountHigh[threadIdx.x] + hist.outlierCountLow[threadIdx.x];
    for (int i = 0; i < numBins; ++i) totalCount += hist.binCount[i][threadIdx.x];

    float ret;
    const auto nf = frac * totalCount;
    // Find the critical bin.
    auto n = hist.outlierCountLow[threadIdx.x];
    if (n > 0 && n >= nf)
    {
        // The precise fractile is in the low-outlier bin.
        // Since this bin is unbounded below, ...
        ret = -inf;
        return ret;
    }

    int i = 0;
    while ((n == 0 || n < nf) && i < numBins)
    {
        n += hist.binCount[i++][threadIdx.x];
    }

    if (n < nf)
    {
        // The precise fractile is in the high-outlier bin.
        // Since this bin is unbounded above, ...
        ret = +inf;
        return ret;
    }

    // Otherwise, the precise fractile is in a normal bin.
    // Interpolate within the critical bin.
    assert(i > 0);
    assert(n >= nf);
    i -= 1;     // Back up to the bin that pushed us over the target.
    auto x0 = hist.lowBound[threadIdx.x] + i * hist.binSize[threadIdx.x];
    const auto ni = hist.binCount[i][threadIdx.x];
    auto m = n - ni;
    assert(m < nf || (m == 0 && nf == 0));
    ret = x0 + hist.binSize[threadIdx.x] * (nf - m) / (ni + 1);

    return ret;
}


// Compute a preliminary scaling factor based on a fractile statistic.
__device__ float PrelimScaleFactor(const ZmwDetectionModel& model,
                                   const DmeEmDevice::LaneHist& hist)
{
    using std::max;  using std::min;
    using std::sqrt;

    // Define a fractile that includes all of the background and half of the pulse frames.
    const auto& bgMode = model.baseline;
    const float& bgVar = bgMode.var;
    const float bgSigma = sqrt(bgVar);
    const float& bgMean = bgMode.mean;
    const float thresh = 2.5f * bgSigma + bgMean;
    const float binSize = hist.binSize[threadIdx.x];

    // Note: This replicates the host original, which never ever included upper
    //       outliers and always includes lower outliers?
    float bgCount = 0;
    float totalCount = hist.outlierCountLow[threadIdx.x];
    {
        int i = 0;
        float binX = hist.lowBound[threadIdx.x];
        float rem = thresh - binX;
        while (rem > binSize)
        {
            totalCount += hist.binCount[i][threadIdx.x];
            binX += hist.binSize[threadIdx.x];
            rem = thresh - binX;
            i++;
        }
        bgCount = totalCount + rem / binSize * hist.binCount[i][threadIdx.x];
        for (; i < numBins; ++i)
        {
            totalCount += hist.binCount[i][threadIdx.x];
        }
    }
    const float fractile = 0.5f * (bgCount + totalCount) / totalCount;

    // Define the scale factor as the ratio between this fractile and the
    // average of the pulse signal means.
    float avgSignalMean = 0.0f;
    assert(model.analogs.size() == numAnalogs);
    for (unsigned int a = 0; a < numAnalogs; ++a)
    {
        avgSignalMean += model.analogs[a].mean;
    }
    avgSignalMean /= static_cast<float>(numAnalogs);

    // Moderate scaling by the clamped model confidence
    const float w = satlin(0, staticConfig.scaleSnrConfTol_, model.confidence);
    auto scaleFactor = Fractile(hist, fractile) / avgSignalMean;
    scaleFactor = (1.0f - w) * scaleFactor + w;

    // Clamp the scale factor to a "reasonable" size
    return min(max(scaleFactor, 0.1f), 10.0f);
}

/// Updates *detModel by increasing the amplitude of all detection modes by
/// \a scale. Also updates all detection mode covariances according
/// to the standard noise model. Ratios of amplitudes among detection modes
/// and properties of the background mode are preserved.
void __device__ ScaleModelSnr(const float& scale,
                              ZmwDetectionModel* detModel)
{
    assert (scale > 0.0f);
    const auto baselineCovar = detModel->baseline.var;
    auto& detectionModes_ = detModel->analogs;
    assert (detectionModes_.size() == numAnalogs);
    for (unsigned int a = 0; a < numAnalogs; ++a)
    {
        auto& dmi = detectionModes_[a];
        dmi.mean *= scale;
        auto cv = Analog(a).excessNoiseCV;
        dmi.var = ModelSignalCovar(cv*cv, dmi.mean, baselineCovar);
    }
    // TODO: Should we update updated_?
}

// Constants
// TODO remove duplication (was from NumericUtil.h)
constexpr float pi_f = 3.1415926536f;

/// The cumulative probability of the standard normal distribution (mean=0,
/// variance=1).
// TODO this duplicated NumericalUtil.h
__device__ float normalStdCdf(float x)
{
    float s = 1.f / sqrt(2.f);
    x *= s;
    const float r = 0.5f * erfc(-x);
    assert((r >= 0.0f) && (r <= 1.0f) || isnan(x));
    return r;
}

/// The cumulative probability at \a x of the normal distribution with \mean
/// and standard deviation \a stdDev.
/// \tparam FP a floating-point numeric type (including m512f).
// TODO this duplicated NumericalUtil.h
__device__ float normalCdf(float x, float mean = 0.0f, float stdDev = 1.0f)
{
    assert(stdDev > 0);
    const float y = (x - mean) / stdDev;
    return normalStdCdf(y);
}

// Apply a G-test significance test to assess goodness of fit of the model
// to the trace histogram.
//
// Note: This is a mostly-complete port of the host version, but it is
//       currently non-functional because it relies on the chi squared
//       distribution which we don't have access to on the GPU.
__device__ GoodnessOfFitTest<float>
Gtest(const DmeEmDevice::LaneHist& histogram, const ZmwDetectionModel& model)
{
    assert(blockDim.x == 64);
    //const auto& bb = histogram.BinBoundaries();
    auto bb = [&](int idx){
        return histogram.lowBound[threadIdx.x] + idx*histogram.binSize[threadIdx.x];
    };
    const auto& bg = model.baseline;

    // Cache the standard deviations.
    const auto bgStdDev = sqrt(bg.var);
    CudaArray<float, numAnalogs> dmStdDev;
    assert(model.analogs.size() == numAnalogs);
    for (unsigned int j = 0; j < numAnalogs; ++j)
    {
        dmStdDev[j] = sqrt(model.analogs[j].var);
    }

    // Compute the bin probabilities according to the model.
    // TODO: Improve accuracy when normalCdf is close to 1.0.
    CudaArray<float, numBins+1> p;
    for (unsigned int i = 0; i < numBins+1; ++i)
    {
        p[i] = Basecaller::normalCdf(bb(i), bg.mean, bgStdDev) * bg.weight;
        for (unsigned int j = 0; j < numAnalogs; ++j)
        {
            const auto& dmj = model.analogs[j];
            p[i] += Basecaller::normalCdf(bb(i), dmj.mean, dmStdDev[j]) * dmj.weight;
        }
    }

    // TODO: By splitting out the first iteration, should be able to fuse the
    // loop in adjacent_difference with the loop above.
    //std::adjacent_difference(p.cbegin(), p.cend(), p.begin());
    float tmp = p[0];
    float diff;
    for (int i = 0; i < numBins; ++i)
    {
        diff = p[i+1] - tmp;
        tmp = p[i+1];
        p[i+1] = diff;
    }

    assert(p[0] >= 0.0f);
    assert(p[0] <= 1.0f);
    assert(p[numBins] >= 0.0f);
    assert(p[numBins] <= 1.0f);

    // Compute the G test statistic.
    const auto n = [&](){
        int n = 0;
        for (int i = 0; i < numBins; ++i)
        {
            n += histogram.binCount[i][threadIdx.x];
        }
        return n;
    }();
    //const auto n = FloatVec(histogram.InRangeCount());
    float g = 0.0f;
    for (int i = 0; i < numBins; ++i)
    {
        const auto obs = histogram.binCount[i][threadIdx.x];
        const auto mod = n * p[i+1];
        const auto t = obs * log(obs/mod);
        if (obs > 0.f) g += t;
    }
    g *= 2.0f;

    // Compute the p-value.
    assert(CoreDMEstimator::nModelParams + 1 < static_cast<unsigned int>(numBins));
    const auto dof = numBins - CoreDMEstimator::nModelParams - 1;
    // TODO disabled because I don't have access to a gpu chi2 distribution on the GPU.
    assert(false);
    const auto pval = 0.f;
    //const auto pval = chi2CdfComp(g * gTestFactor_, dof);

    return {g, static_cast<float>(dof), pval};
}

// Compute the confidence factors of a model estimate, given the
// diagnostics of the estimation, a reference model.
__device__ PacBio::Cuda::Utility::CudaArray<float, ConfFactor::NUM_CONF_FACTORS>
ComputeConfidence(const DmeDiagnostics<float>& dmeDx,
                  const ZmwDetectionModel& refModel,
                  const ZmwDetectionModel& modelEst)
{
    const auto mldx = dmeDx.mldx;
    CudaArray<float, ConfFactor::NUM_CONF_FACTORS> cf;
    for (auto& val : cf) val = 1.f;

    // Check EM convergence.
    cf[ConfFactor::CONVERGED] = mldx.converged;

    // Check for missing baseline component.
    const auto& bg = modelEst.baseline;
    // TODO: Make this configurable.
    // Threshold level for background fraction.
    static const float bgFracThresh0 = 0.05f;
    static const float bgFracThresh1 = 0.15f;
    float x = satlin(bgFracThresh0, bgFracThresh1, bg.weight);
    cf[ConfFactor::BL_FRACTION] = x;

    // Check magnitude of residual baseline mean.
    x = bg.mean * bg.mean / bg.var;
    // TODO: Make this configurable.
    static const float bgMeanTol = 1.0f;
    assert(bgMeanTol > 0.0f);
    x = exp(-x / (2*bgMeanTol*bgMeanTol));
    cf[ConfFactor::BL_CV] = x;

    // Check for large deviation of baseline variance from reference variance.
    const auto& refBgVar = refModel.baseline.var;
    x = log2(bg.var / refBgVar);
    // TODO: Make this configurable.
    const float bgVarTol = 1.5f / (0.5f + refModel.confidence);
    x = exp(-x*x / (2*bgVarTol*bgVarTol));
    cf[ConfFactor::BL_VAR_STABLE] = x;

    // Check for missing pulse components.
    // Require that the first (brightest) and last (dimmest) are not absent.
    x = 1.0f;
    const auto& detModes = modelEst.analogs;
    const float analogMixFracThresh1_ = staticConfig.analogMixFracThresh1_;
    const float analogMixFracThresh0_ = staticConfig.analogMixFracThresh0_;
    if (analogMixFracThresh1_ > 0.0f)
    {
        assert(detModes.size() >= 1);
        assert(analogMixFracThresh0_ < analogMixFracThresh1_);
        x *= satlin(analogMixFracThresh0_, analogMixFracThresh1_, detModes.front().weight);
        x *= satlin(analogMixFracThresh0_, analogMixFracThresh1_, detModes.back().weight);
    }
    cf[ConfFactor::ANALOG_REP] = x;

    // Check for low SNR.
    x = detModes.back().mean;  // Assumes last analog is dimmest.
    const auto bgSigma = sqrt(bg.var);
    x /= bgSigma;
    x = satlin(staticConfig.snrThresh0_, staticConfig.snrThresh1_, x);
    cf[ConfFactor::SNR_SUFFICIENT] = x;

    // Check for large decrease in SNR.
    // This factor is specifically designed to catch registration errors in the
    // fit when the brightest analog is absent. In such cases, the weight of
    // the dimmest fraction can be substantial (the fit presumably robs some
    // weight from the background component)
    if (staticConfig.snrDropThresh_ < 0.0f) cf[ConfFactor::SNR_DROP] = 1.0f;
    else
    {
        const auto snrEst = detModes[0].mean / bgSigma;
        const auto& refDetModes = refModel.analogs;
        assert(refDetModes.size() >= 2);
        const auto& refSignal0 = refDetModes[0].mean;
        //PBAssert(all(refSignal0 >= 0.0f), "Bad SignalMean.");
        const auto& refSignal1 = refDetModes[1].mean;
        //PBAssert(all(refSignal1 >= 0.0f), "Bad SignalMean.");
        const auto refBgSigma = sqrt(refModel.baseline.var);
        //PBAssert(all(refBgSigma >= 0.0f), "Bad baseline sigma.");
        const auto refSnr0 = refSignal0 / refBgSigma;
        auto refSnr1 = refSignal1 / refBgSigma;
        refSnr1 *= staticConfig.snrDropThresh_;
        refSnr1 *= min(refModel.confidence, 1.0f);
        //PBAssert(all(refSnr1 < refSnr0),
        //         "Bad threshold in SNR Drop confidence factor.");
        x = satlin(refSnr1, sqrt(refSnr0*refSnr1), snrEst);
        cf[ConfFactor::SNR_DROP] = x;
    }

    // The G-test as a goodness-of-fit score.
    cf[ConfFactor::G_TEST] = dmeDx.gTest.pValue;

    return cf;
}

// TODO this code replicates host stat accumulator for float and PBHalf2 below
__device__ float Mean(const StatAccumState& stats)
{
    float mean = stats.moment1[threadIdx.x] / stats.moment0[threadIdx.x];
    return mean + stats.offset[threadIdx.x];
};

/// The unbiased sample variance of the aggregated samples.
/// NaN if Count() < 2.
__device__ float Variance(const StatAccumState& stats)
{
    float var = stats.moment1[threadIdx.x] * stats.moment1[threadIdx.x];
    var /= stats.moment0[threadIdx.x];
    var = (stats.moment2[threadIdx.x] - var);
    var /= (stats.moment0[threadIdx.x] - 1.0f);
    var = max(var, 0.0f);

    const float nan = std::numeric_limits<float>::quiet_NaN();
    return stats.moment0[threadIdx.x] > 1.0f ? var : nan;
};

__device__ void EvolveModel(FiTypeDevice estFI,
                            const BaselinerStatAccumState& blStats,
                            ZmwDetectionModel* model,
                            FiTypeDevice modFI)
{
    // Current and next model
    auto mod0Cntr = modFI.lo + modFI.up / 2;
    auto mod1Cntr = estFI.lo + estFI.up / 2;

    const float thl = 0.5f * (mod0Cntr + mod1Cntr);

    // TODO: Make these configurable.       ALP: Adaptive Laser Power
    static const float t00 = 56160.0f;   // 50th percentile of ALP duration
    static const float t10 = 168480.0f;  // 97.5th %-ile of ALP duration

    // Compute the nominal confidence half-life.
    const float& hl0 = staticConfig.confidHalfLife0_;
    const float& hl1 = staticConfig.confidHalfLife1_;
    const float hl = hl0 + (hl1 - hl0) * satlin(t00, t10, thl);     // frames

    // EvolveConfidence
    const auto t01 = mod0Cntr;
    const auto t11 = mod1Cntr;
    auto confHalfLife = hl;
    const float tDiff = static_cast<float>(t11 - t01);
    const auto m = exp2(-abs(tDiff)/confHalfLife);
    model->confidence *= m;
}

__device__ void PrelimEstimate(const BaselinerStatAccumState& blStatAccState,
                               ZmwDetectionModel* model)
{
    assert(model != nullptr);

    const float nBlFrames   = blStatAccState.baselineStats.moment0[threadIdx.x];
    const float totalFrames = blStatAccState.fullAutocorrState.basicStats.moment0[threadIdx.x];
    const float blWeight    = max(nBlFrames / totalFrames, 0.01f);

    // Reject baseline statistics with insufficient data
    float nBaselineMin(3.0f);
    auto mask = nBlFrames >= nBaselineMin;
    ZmwAnalogMode& m0blm = model->baseline;
    const StatAccumState& blsa  = blStatAccState.baselineStats;

    auto blMean = mask ? Mean(blsa)     : m0blm.mean;
    auto blVar  = mask ? Variance(blsa) : m0blm.var;

    const auto blVarMax = baseConfig.BaselineVarianceMax();
    blVar = std::clamp(blVar, baseConfig.BaselineVarianceMin(), blVarMax);

    auto& detectionModes = model->analogs;

    assert(isfinite(blMean));
    assert(isfinite(blVar) && (blVar > 0.0f));
    assert (detectionModes.size() == numAnalogs);

    // Rescale
    auto scale = sqrt(blVar / m0blm.var);

    for (uint32_t i = 0; i < numAnalogs; ++i)
    {
        auto& mode = detectionModes[i];
        mode.mean = mode.mean * scale;
        auto cv = Analog(i).excessNoiseCV;
        mode.var  = ModelSignalCovar(cv*cv, mode.mean, blVar);
        mode.weight = 0.25f*(1.0f-blWeight);
    }

    model->baseline.mean = blMean;
    model->baseline.var = blVar;
    model->baseline.weight = blWeight;

    // Frame interval is not updated since it is not exported

    float conf = 0.1f * satlin(0.0f, 500.0f, nBlFrames - nBaselineMin);
    conf *= satlin(blVarMax, 0.5f * (baseConfig.BaselineVarianceNominal() + blVarMax), blVar);
    model->confidence = conf;
}

// Use the trace histogram and the input detection model to compute a new
// estimate for the detection model. Mix the new estimate with the input
// model, weighted by confidence scores. That result is returned in detModel.
__device__ void EstimateLaneDetModel(FiTypeDevice estFI,
                                     const DmeEmDevice::LaneHist& hist,
                                     const BaselinerStatAccumState& blStatAccState,
                                     LaneDetModel* detModel,
                                     FiTypeDevice modFI)
{
    assert(detModel != nullptr);

    ZmwDetectionModel model0;
    if (threadIdx.x%2 == 0)
        model0.Assign<0>(*detModel, threadIdx.x/2);
    else
        model0.Assign<1>(*detModel, threadIdx.x/2);

    FiTypeDevice wmFI;
    EvolveModel(estFI, blStatAccState, &model0, modFI);
    modFI = estFI;

    // Update model based on estimate of baseline variance
    // with confidence-weighted method
    ZmwDetectionModel workModel = model0;
    wmFI = modFI;

    PrelimEstimate(blStatAccState, &workModel);

    UpdateModel(workModel, &model0);
    modFI = wmFI;

    // Make a working copy of the detection model.
    workModel = model0;
    wmFI = modFI;

    // The term "mode" refers to a component of the mixture model.
    auto& bgMode = workModel.baseline;
    auto& pulseModes = workModel.analogs;

    const auto& numFrames = TotalCount(hist);

    // Scale the model based on fractile of the data.
    const auto scaleFactor = PrelimScaleFactor(workModel, hist);
    ScaleModelSnr(scaleFactor, &workModel);

    const auto binSize = hist.binSize[threadIdx.x];
    const auto hBinSize = 0.5f * binSize;

    // Define working variables for model parameters.
    const auto nModes = numAnalogs + 1;
    using ModeArray = CudaArray<float, nModes>;
    ModeArray rho;     // Mixture fraction for each mode.
    ModeArray mu;      // Mean of each mode.
    ModeArray var;     // Variance of each mode.

    const float rhoMin = 0.001f;
    const float blVarMin = baseConfig.BaselineVarianceMin();
    const float blVarMax = baseConfig.BaselineVarianceMax();

    // Variance associated with data binning.
    const auto varQuant = binSize * binSize / 12.f;

    rho[0] = max(bgMode.weight, rhoMin);
    assert(rho[0] <= 1.0f);
    mu[0] = bgMode.mean;
    var[0] = bgMode.var + varQuant;
    assert(var[0] >= blVarMin);
    for (unsigned int a = 0; a < numAnalogs; ++a)
    {
        const auto k = a + 1;
        //PBAssert(k < nModes, "k < nModes");
        const auto& pma = pulseModes[a];
        rho[k] = pma.weight;
        assert(rho[k] <= 1.0f);
        mu[k] = pma.mean;
        var[k] = pma.var + varQuant;
        assert(var[k] >= blVarMin);
    }

    // Enforce sanity bound on baseline variance.
    var[0] = min(var[0], blVarMax);

    // Enforce normalization of mixture fractions.
    {
        float rhoSum = 0.f;
        for (int i = 0; i < numAnalogs+1; ++i)
        {
            rhoSum += rho[i];
        }
        auto rhoSumInv = 1.f / rhoSum;
        for (int i = 0; i < numAnalogs+1; ++i)
        {
            rho[i] *= rhoSumInv;
        }
    }

    // Initialize estimates to initial model.
    ModeArray rhoEst = rho;
    ModeArray muEst = mu;
    ModeArray varEst = var;

    // The relative pulse amplitudes.
    CudaArray<float, numAnalogs> rpa;
    for (unsigned int a = 0; a < numAnalogs; ++a)
    {
        rpa[a] = Analog(a).relAmplitude;
        // TODO data error handling
        //if (rpa[a] <= 0.0f)
        //{
        //    throw PBException("Bad relative amplitude in analog properties."
        //                      " Relative amplitudes must be greater than zero.");
        //}
    }

    // Initialize the model parameter for pulse amplitude scale, which is
    // needed for the prior when calculating the posterior.
    float s = [&]()
    {
        float tmp1 = 0.f;
        float tmp2 = 0.f;
        for (int i = 0; i < numAnalogs; ++i)
        {
            tmp1 += mu[i+1]*rpa[i];
            tmp2 += rpa[i]*rpa[i];
        }
        return tmp1 / tmp2;
    }();
    //FloatVec s = (mu.tail(numAnalogs) * rpa.cast<FloatVec>()).sum() / rpa.square().sum();

    // Prior hyperparameters for scale parameter.
    // Undo the preliminary scaling based on percentile to get back to the
    // means of initModel.
    const float sExpect = s / scaleFactor;

    // sExpectWeight is the inverse of the variance of the normal prior for s.
    const float sExpectWeight = model0.confidence * staticConfig.pulseAmpRegCoeff_;

    // Log likelihood--really a posterior since we've added a prior for s.
    float logLike {numeric_limits<float>::lowest()};
    float logLikePrev {logLike};

    // Iteration limit for EM.
    const unsigned int iterLimit = staticConfig.emIterLimit_;

    // Define lower bound for bin probability.
    const float binProbMin
            = max(numFrames * binSize, 1.0f)
            * numeric_limits<float>::min();

    // Initialize intra-lane failure codes.
    int32_t zStatus = DmeEmDevice::OK;
    if (numFrames < CoreDMEstimator::nFramesMin) zStatus |= DmeEmDevice::INSUF_DATA;

    DmeDiagnostics<float> dmeDx {};
    dmeDx.fullEstimation = true;

    // TODO: Need to track frame interval.
//    dmeDx.startFrame = dtbs.front()->StartFrame();
//    dmeDx.stopFrame = dtbs.back()->StopFrame();

    MaxLikelihoodDiagnostics<float>& mldx = dmeDx.mldx;
    mldx.degOfFreedom = numFrames - CoreDMEstimator::nModelParams;

    // Expectation-maximization of grouped data.
    // See G. J. McLachlan and P. N. Jones,
    // Biometrics, Vol. 44, No. 2 (June, 1988), pp. 571-578.
    // http://www.jstor.org/stable/2531869

    const float n_j_sum = [&](){
        float sum = 0;
        for (int i = 0; i < numBins; ++i)
        {
            sum += hist.binCount[i][threadIdx.x];
        }
        return sum;
    }();

    unsigned int it = 0;
    for (; it < iterLimit; ++it)
    {
        // E-step

        // Inverses of variances.
        ModeArray hVarinv = var;
        for (auto& val : hVarinv) val = 0.5f / val;

        const float log_2pi = log(2.0f * pi_f);
        ModeArray prefactors;
        ModeArray c_i;
        ModeArray mom1;
        for (int i = 0; i < rho.size(); ++i)
        {
            prefactors[i] = log(rho[i]) - 0.5f * log(var[i]) - 0.5f * log_2pi;
            c_i[i] = 0.f;
            mom1[i] = 0.f;
        }

        float probSum(0.0f);
        float weightedProbSum(0.0f);
        float mom2 = 0.0f;   // Only needed for background mode.
        // correction terms to account for the fact that we'll use the
        // current mu in the loop below, while the math really wants
        // the updated mu not computed until later.
        float correct1 = 0.0f;
        float correct2 = 0.0f;

        struct CornerVals
        {
            float tau[nModes];
            float cProb;
        };
        auto cornerComp = [&](int b, CornerVals& cv, float x)
        {
            // First compute the log of the component probabilities.
            for (int i = 0; i < nModes; ++i)
            {
                const auto& y = x - mu[i];
                cv.tau[i] = prefactors[i] - y*y*hVarinv[i];
            }

            // Next compute the log likelihood of each bin boundary, which is
            // the log of the sum of exp(tau_i_x) over modes (i).
            // Use log-sum-exp trick to avoid uniform underflow. (PTVF2007, Equation 16.1.9)
            float tauMax = cv.tau[0];
            for (int i = 1; i < nModes; ++i)
            {
                tauMax = max(tauMax, cv.tau[i]);
            }
            float llb = [&](){
                float ret = 0.f;
                for (int i = 0; i < nModes; ++i)
                {
                    ret += __expf(cv.tau[i] - tauMax);
                }
                return __logf(ret) + tauMax;
            }();

            // Convert to likelihood ratio.
            for (int i = 0; i < nModes; ++i)
            {
                cv.tau[i] = __expf(cv.tau[i] - llb);
            }

            // Record the total probability density at the bin boundaries.
            cv.cProb = __expf(llb);
        };

        auto centerComp = [&](int b, const CornerVals& c1, const CornerVals& c2, float x0, float x1)
        {
            // Constrain bin probability to a minimum positive value.
            // Avoids 0 * log(0) in computation of log likelihood.
            auto binProb = max(hBinSize * (c1.cProb + c2.cProb), binProbMin);
            float binCount = hist.binCount[b][threadIdx.x];
            probSum += binProb;
            weightedProbSum += binCount * __logf(binProb);

            const auto factor = binCount / binProb;

            for (unsigned int m = 0; m < nModes; ++m)
            {
                // Relative weight of each mode. CSMM2002, Equation 5.
                // Use trapezoidal approximation of expectation of tau over each bin.
                auto tmp1 = c1.cProb * c1.tau[m];
                auto tmp2 = c2.cProb * c2.tau[m];

                c_i[m] += (tmp1 + tmp2) * factor;
                mom1[m] += (tmp1 * x0 + tmp2 * x1) * factor;
            }

            x0 -= mu[0];
            x1 -= mu[0];

            float co2 = c1.cProb * c1.tau[0];
            float co1 = co2 * x0;
            float m = x0 * co1;

            float tmp = c2.cProb * c2.tau[0];
            co2 += tmp;
            tmp *= x1;
            co1 += tmp;
            m += x1 * tmp;

            tmp = factor;
            mom2 += m * tmp;
            correct2 += co2 * tmp;
            correct1 += co1 * tmp;
        };

        // TODO: Check for bins where binProb == 0 but n_j > 0.
        CornerVals c1;
        CornerVals c2;
        float x0 = hist.lowBound[threadIdx.x];
        float x1 = x0 + binSize;
        cornerComp(0, c1, x0);
        for (unsigned int b = 0; b < numBins; ++b)
        {
            cornerComp(b+1, c2, x1);
            centerComp(b, c1, c2, x0, x1);
            c1 = c2;
            x0 = x1;
            x1 += binSize;
        }
        // TODO: Note the cancellation of the (0.5f * binSize) factor. Possible minor optimization opportunity.
        // TODO: When used, tau is always multiplied by boundaryProb. So why bother dividing that factor out when first computing tau?

        for (int i = 0; i < nModes; ++i)
        {
            c_i[i] *= 0.5f * binSize;
            mom1[i] *= 0.5f * binSize;
        }

        // Update log posterior.
        logLikePrev = logLike;
        logLike = weightedProbSum - n_j_sum * log(probSum);
        auto sdiff = s-sExpect;
        logLike -= 0.5f * sdiff * sdiff * sExpectWeight;

        // Check for convergence.
        const float deltaLogLike {logLike - logLikePrev};
        const float convTol = 1e-4f * abs(logLike);  // TODO: Make this configurable.
        // TODO if this loop body ever gets any code that does anything, may need to
        // use warp voting to share information for the whole lane?
        if (deltaLogLike < -convTol)
        {
            // TODO: We need lane tracking.
            // Ideally, this should not happen.
            // TODO: Should we abort estimation for these ZMWs?
            // TODO: PBLOGGER_DEBUG is broken.
//            PBLOGGER_DEBUG(logger_)
//                    << "Likelihood decreased during EM: Pool " << poolId_
//                    << ", iteration " << it << '.';
        }

        // TODO: Seems like we ought to eliminate or relax the lower bound
        // condition on deltaLogLike here.
        bool conv = (deltaLogLike >= 0) && (deltaLogLike < convTol)
                  && (logLike >= mldx.logLike);
        mldx.Converged(conv, it, logLike, deltaLogLike);

        // Update result for converged ZMWs.
        if (conv) for (unsigned int k = 0; k < nModes; ++k)
        {
            rhoEst[k] = rho[k];
            muEst[k]  = mu[k];
            varEst[k] = var[k];
        }

        // Note: This __all_sync is a bit weird.  It was put in place to match the host
        //       impl where the SIMD approach required all ZMW in a lane to converge if
        //       we're going to break early.  However here:
        //       * This sync is only for a warp, which really is only 32 ZMW not 64, so
        //         we're already inconsistent
        //       * With the CUDA SIMT model we could have threads break
        //         and exit individually when they are done with no
        //       Ideally we'd match perfectly and do a whole lane-level synchronization,
        //       but that's nontrivial to write since it involves coordination between
        //       two warps
        if (!staticConfig.iterToLimit_ && __all_sync(0xFFFFFFFF, mldx.converged || (zStatus != static_cast<int>(DmeEmDevice::OK))))
        {
            // TODO: Maybe count how many times we achieve this?
            break;
        }

        // M-step

        // Mixing fractions.
        // Constrain each component to have some minimal mixing fraction.
        float rhoSum = 0.0f;
        for (int i = 0; i < nModes; ++i)
        {
            c_i[i] = max(c_i[i], 0.1f);
            rho[i] = c_i[i] / n_j_sum;
            rhoSum += rho[i];
        }
        for (auto& x : rho) x /= rhoSum;

        auto oldMu = mu[0];
        // Background mean.
        mu[0] = mom1[0] / c_i[0];
        auto muDiff = oldMu - mu[0];

        // Amplitude scale parameter.
        float numer = 0.0f;
        float denom = 0.0f;
        for (unsigned int a = 0; a < numAnalogs; ++a)
        {
            const auto i = a + 1;
            auto varinv = 2.0f * hVarinv[i];
            numer += rpa[a] * varinv * mom1[i];
            denom += rpa[a] * rpa[a] * varinv * c_i[i];
        }

        // Add regularization/prior terms.
        numer += sExpect * sExpectWeight;
        denom += sExpectWeight;

        s = numer / denom;

        // The minimum bound for the pulse-amplitude scale parameter.
        static const float minSnr = 0.5f;
        const float minPulseMean = max(1.0f, baseConfig.signalScaler);
        auto rpaMin = rpa[0]; for (int i = 1; i < rpa.size(); ++i) rpaMin = min(rpaMin, rpa[i]);
        const auto sMin = max(minSnr * sqrt(var[0]), minPulseMean) / rpaMin;

        // Constrain s > sMin.
        const auto veryLowSignal = (s < sMin);
        if (veryLowSignal)
        {
            s = sMin;
            zStatus |= DmeEmDevice::VLOW_SIGNAL;
        }

        // Pulse mode means.
        for (int i = 0; i < numAnalogs; ++i) mu[i+1] = s * static_cast<float>(rpa[i]);

        // Background variance.
        // Need to apply correction terms since it was computed above with the old
        // mean instead of the newest update.
        // So far we've only run on "friendly" data, in which case it appears
        // that these correction terms make no practical difference.  Their
        // utility needs to be evaluated on more realistic data, as we will
        // run measurably faster if we can skip the computation of one or even
        // both of these terms.
        mom2 += muDiff * correct1 + muDiff * muDiff * correct2;
        mom2 *= hBinSize;
        var[0] = mom2 / c_i[0] + varQuant;
        var[0] = std::clamp(var[0], blVarMin, blVarMax);

        // Each pulse mode variance is computed as a function of the background
        // variance and the pulse mode mean.
        // Note that we've ignored these dependencies when updating the means
        // and the background variance above.
        for (unsigned int i = 0; i < numAnalogs; ++i)
        {
            auto cv = Analog(i).excessNoiseCV;
            var[i+1] = ModelSignalCovar(cv*cv, mu[i+1], var[0]);
        }
    }

    if (!mldx.converged) zStatus |= DmeEmDevice::NO_CONVERGE;

    using std::isfinite;

    // Package estimation results.
    //PBAssert(all(isfinite(rhoEst[0])), "all(isfinite(rhoEst[0])");
    bgMode.weight = rhoEst[0];
    //PBAssert(all(isfinite(muEst[0])), "all(isfinite(muEst[0]))");
    bgMode.mean = muEst[0];
    //PBAssert(all(isfinite(varEst[0])), "all(isfinite(varEst[0]))");
    bgMode.var = varEst[0];
    for (unsigned int a = 0; a < numAnalogs; ++a)
    {
        auto& pda = pulseModes[a];
        const auto i = a + 1;
        //PBAssert(all(isfinite(rhoEst[i])), "all(isfinite(rhoEst[i])");
        pda.weight = rhoEst[i];
        //PBAssert(all(isfinite(muEst[i])), "all(isfinite(muEst[i]))");
        pda.mean = muEst[i];
        //PBAssert(all(isfinite(varEst[i])), "all(isfinite(varEst[i]))");
        pda.var = varEst[i];
    }

    // Note: this is disabled until we have a chi squared cfd on the gpu.
    assert (staticConfig.gTestFactor_ < 0.);
    if (staticConfig.gTestFactor_ >= 0.0f) dmeDx.gTest = Gtest(hist, workModel);
    else assert(dmeDx.gTest.pValue == 1.0f);

    // Compute confidence score.
    dmeDx.confidFactors = ComputeConfidence(dmeDx, model0, workModel);
    {
        using std::min;  using std::max;
        float conf = 1.0f;
        for (const auto& cf : dmeDx.confidFactors) conf *= cf;
        conf = min(max(0.0f, conf), 1.0f);
        if (conf < staticConfig.successConfThresh_) conf = 0;
        workModel.confidence = conf;
    }
    // TODO: Push results to DmeDumpCollector.
    //    if (this->dmeDumpCollector_)
    //    {
    //        // TODO: Add lane and zmw event codes.
    //        this->dmeDumpCollector_->CollectRawEstimate1D(*dtbs.front(), hist, dmeDx, *workModel);
    //    }

    // Update the current model, model0.
    UpdateModel(workModel, &model0);
    modFI = wmFI;

    // Transcribe back to *detModel.
    model0.Export(threadIdx.x/2, threadIdx.x%2, detModel);
}

__global__ void EstimateKernel(FiTypeDevice estFI,
                               Cuda::Memory::DeviceView<const DmeEmDevice::LaneHist> hists,
                               Cuda::Memory::DeviceView<const BaselinerStatAccumState> blStatsState,
                               Cuda::Memory::DeviceView<LaneDetModel> models,
                               FiTypeDevice currFI)
{
    EstimateLaneDetModel(estFI, hists[blockIdx.x], blStatsState[blockIdx.x], &models[blockIdx.x], currFI);
}


void DmeEmDevice::EstimateImpl(const PoolHist &hist, 
                               const Data::BaselinerMetrics& metrics,
                               PoolDetModel *detModelPool) const
{
    // We assume that frame intervals of the trace histogram and the baseliner
    // statistics are equal.
    const auto& hfi = hist.frameInterval;
    assert(hfi == metrics.frameInterval);

    const auto& pfi = detModelPool->frameInterval;

    FiTypeDevice hstFI { hfi.Lower(), hfi.Upper() };
    FiTypeDevice modFI { pfi.Lower(), pfi.Upper() };

    Cuda::PBLauncher(EstimateKernel, hist.data.Size(), laneSize)
                    (hstFI, hist.data, metrics.baselinerStats, detModelPool->data, modFI);
    Cuda::CudaSynchronizeDefaultStream();

    detModelPool->frameInterval = hfi;
}

__global__ void InitModel(Cuda::Memory::DeviceView<const BaselinerStatAccumState> stats,
                          Cuda::Memory::DeviceView<LaneDetModel> models)
{
    auto& blStats = stats[blockIdx.x];
    auto& model = models[blockIdx.x];

    const auto& blsa = blStats.baselineStats;
    const auto& basicStats = blStats.fullAutocorrState.basicStats;

    // TODO this code replicates host stat accumulator
    auto half2Mean = [](const auto& stats) -> PBHalf2
    {
        return { stats.moment1[threadIdx.x*2]/stats.moment0[threadIdx.x*2],
                 stats.moment1[threadIdx.x*2+1]/stats.moment0[threadIdx.x*2+1]
        };
    };

    /// The unbiased sample variance of the aggregated samples.
    /// NaN if Count() < 2.
    auto half2Variance = [](const auto& stats) -> PBHalf2
    {
        const PBHalf2 nan = std::numeric_limits<float>::quiet_NaN();
        const PBHalf2 mom0 { stats.moment0[2*threadIdx.x], stats.moment0[2*threadIdx.x+1]};
        const PBFloat2 mom1 { stats.moment1[2*threadIdx.x], stats.moment1[2*threadIdx.x+1]};
        const PBFloat2 mom2 { stats.moment2[2*threadIdx.x], stats.moment2[2*threadIdx.x+1]};

        PBFloat2 tmp = mom1 * mom1 / mom0;
        tmp = (mom2 - tmp) / (mom0 - 1.0f);
        // We're having problems with the baseline variance overflowing a half precision
        // storage.  Something is already terribly wrong if our variance is over
        // 65k, but we'll put a limiter here because having a literal infinity run
        // around is causing problems elsewhere
        PBHalf2 var = clamp(PBHalf2{tmp.X(), tmp.Y()}, 0.0f, 65000.0f);
        return Blend(mom0 > 1.0f, var, nan);
    };

    PBHalf2 blMean =  half2Mean(blsa);
    PBHalf2 blVar  = half2Variance(blsa);
    const PBHalf2 blWeight = {
        blsa.moment0[2*threadIdx.x] / basicStats.moment0[2*threadIdx.x],
        blsa.moment0[2*threadIdx.x+1] / basicStats.moment0[2*threadIdx.x+1]
    };

    // Set the confidence to a small nominal value.
    model.Confidence()[threadIdx.x] = 0.1f;

    // If baseline frame count is insufficient, blMean and blVar can be NaN.
    // In this case, just adopt some semi-arbitrary fallback value.
    blMean = Blend(isnan(blMean), baseConfig.fallbackBaselineMean, blMean);
    blVar = Blend(isnan(blVar), baseConfig.fallbackBaselineVariance, blVar);

    // Constrain variance to reasonable range.
    const float blVarMin = baseConfig.BaselineVarianceMin();
    const float blVarMax = baseConfig.BaselineVarianceMax();
    blVar = clamp(blVar, blVarMin, blVarMax);


    const auto refSignal = staticConfig.refSnr_ * sqrt(blVar);
    const auto& aWeight = 0.25f * (1.0f - blWeight);
    model.BaselineMode().means[threadIdx.x] = blMean;
    model.BaselineMode().vars[threadIdx.x] = blVar;
    model.BaselineMode().weights[threadIdx.x] = blWeight;
    for (int a = 0; a < numAnalogs; ++a)
    {
        const auto aMean = max(blMean + staticConfig.analogs[a].relAmplitude * refSignal, 0);
        auto& aMode = model.AnalogMode(a);
        aMode.means[threadIdx.x] = aMean;

        // This noise model assumes that the trace data have been converted to
        // photoelectron units.
        auto cv = Analog(a).excessNoiseCV;
        aMode.vars[threadIdx.x] = min(65000.0f, ModelSignalCovar(cv*cv, aMean, blVar));

        aMode.weights[threadIdx.x] = aWeight;
    }
}

DmeEmDevice::PoolDetModel
DmeEmDevice::InitDetectionModels(const PoolBaselineStats& blStats) const
{
    PoolDetModel pdm (PoolSize(), Cuda::Memory::SyncDirection::HostReadDeviceWrite, SOURCE_MARKER());

    Cuda::PBLauncher(InitModel, PoolSize(), laneSize/2)(blStats.baselinerStats, pdm.data);
    Cuda::CudaSynchronizeDefaultStream();

    pdm.frameInterval = blStats.frameInterval;
    return pdm;
}

}}}     // namespace PacBio::Mongo::Basecaller
