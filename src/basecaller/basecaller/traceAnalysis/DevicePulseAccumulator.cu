#include "hip/hip_runtime.h"
//  Copyright (c) 2019, Pacific Biosciences of California, Inc.
//
//  All rights reserved.
//
//  Redistribution and use in source and binary forms, with or without
//  modification, are permitted provided that the following conditions are met:
//  * Redistributions of source code must retain the above copyright
//    notice, this list of conditions and the following disclaimer.
//  * Redistributions in binary form must reproduce the above copyright
//    notice, this list of conditions and the following disclaimer in the
//    documentation and/or other materials provided with the distribution.
//  * Neither the name of Pacific Biosciences nor the names of its
//    contributors may be used to endorse or promote products derived from
//    this software without specific prior written permission.
//
//  NO EXPRESS OR IMPLIED LICENSES TO ANY PARTY'S PATENT RIGHTS ARE GRANTED BY
//  THIS LICENSE.  THIS SOFTWARE IS PROVIDED BY PACIFIC BIOSCIENCES AND ITS
//  CONTRIBUTORS "AS IS" AND ANY EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT
//  LIMITED TO, THE IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A
//  PARTICULAR PURPOSE ARE DISCLAIMED. IN NO EVENT SHALL PACIFIC BIOSCIENCES OR
//  ITS CONTRIBUTORS BE LIABLE FOR ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL,
//  EXEMPLARY, OR CONSEQUENTIAL DAMAGES (INCLUDING, BUT NOT LIMITED TO,
//  PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES; LOSS OF USE, DATA, OR PROFITS; OR
//  BUSINESS INTERRUPTION) HOWEVER CAUSED AND ON ANY THEORY OF LIABILITY, WHETHER
//  IN CONTRACT, STRICT LIABILITY, OR TORT (INCLUDING NEGLIGENCE OR OTHERWISE)
//  ARISING IN ANY WAY OUT OF THE USE OF THIS SOFTWARE, EVEN IF ADVISED OF THE
//  POSSIBILITY OF SUCH DAMAGE.

#include <basecaller/traceAnalysis/DevicePulseAccumulator.h>

#include <dataTypes/BatchData.cuh>
#include <dataTypes/BatchVectors.cuh>

#include <common/cuda/memory/DeviceOnlyArray.cuh>
#include <common/cuda/PBCudaSimd.cuh>
#include <common/MongoConstants.h>

using namespace PacBio::Cuda;
using namespace PacBio::Cuda::Memory;
using namespace PacBio::Cuda::Utility;
using namespace PacBio::Mongo::Data;

namespace PacBio {
namespace Mongo {
namespace Basecaller {

namespace {

// I don't want to promote this overload to a broader scope until I'm sure we need it
// I don't know if we want a full PBUInt2 type added, or if using raw uint2 is
// preferrable, or if using a pair of ints is just an implementation quirk of this
// particular file
inline __device__ uint2 Blend(PBBool2 cond, uint l, uint2 r) {
    uint2 ret;
    ret.x = cond.X() ? l : r.x;
    ret.y = cond.Y() ? l : r.y;
    return ret;
};


template <size_t blockThreads>
class Segment
{
public:
    __device__  Segment()
    {
        for (int i = 0; i < blockThreads; ++i)
        {
            startFrame_[i] = make_uint2(0,0);
            endFrame_[i] = make_uint2(0,0);
            signalFrstFrame_[i] = PBShort2(0);
            signalLastFrame_[i] = PBShort2(0);
            signalMax_[i] = PBShort2(0);
            signalTotal_[i] = PBShort2(0);
            signalM2_[i] = PBHalf2(0.0f);
            label_[i] = PBShort2(0);
        }
    }

    static __device__ PBBool2 IsPulseUpState(PBShort2 label)
    {
        return (numAnalogs < label) && (label <= 2*numAnalogs);
    }

    static __device__ PBBool2 IsPulseDownState(PBShort2 label)
    {
        return (2*numAnalogs < label);
    }

    __device__ PBBool2 IsNewSegment(PBShort2 label) const
    {
        return IsPulseUpState(label) || ((label == 0) && (label_[threadIdx.x] != 0));
    }

    __device__ PBBool2 IsPulse() const
    {
        return label_[threadIdx.x] != 0;
    }


    __device__ PBShort2 FullFrameLabel() const
    {
        PBShort2 ret = label_[threadIdx.x];
        ret = Blend(IsPulseDownState(ret), ret - 2*numAnalogs, ret);
        ret = Blend(IsPulseUpState(ret), ret - numAnalogs, ret);
        return ret;
    }

    template <int id>
    __device__ Data::Pulse ToPulse(uint32_t frameIndex)
    {
        static_assert(id < 2 && id >= 0, "Invalid index");
        using NucleotideLabel = Data::Pulse::NucleotideLabel;

        // This is potentially a short term hack.  The types like PBShort2 have
        // a Get function to handle getting the 0th or 1st element.  It would be
        // nice to handle this semmetrically, but right now we don't have a
        // PBUint2 type, and I don't think this is enough motivation to add one
        auto Get = [](uint2& var) -> uint& { return id ? var.x : var.y; };

        Data::Pulse ret;
        Get(endFrame_[threadIdx.x]) = frameIndex;
        auto start = Get(startFrame_[threadIdx.x]);
        short width = frameIndex - start;

        auto raw_mean = PBHalf2(signalTotal_[threadIdx.x] + signalLastFrame_[threadIdx.x] + signalFrstFrame_[threadIdx.x]) / width;
        auto raw_mid = PBHalf2(signalTotal_[threadIdx.x]) / (width - 2);

        const auto maxSignal = Data::Pulse::SignalMax();


        // TODO: This is a hard coded analog mapping that really needs to be handed in somehow
        auto LabelToAnalog = [](short label) {
            switch (label)
            {
            case 0:
                return NucleotideLabel::NONE;
            case 1:
                return NucleotideLabel::A;
            case 2:
                return NucleotideLabel::C;
            case 3:
                return NucleotideLabel::G;
            default :
            assert(label == 4);
                return NucleotideLabel::T;
            }
        };

        ret.Start(start)
            .Width(width)
            .MeanSignal(min(maxSignal, max(0.0f, raw_mean.Get<id>())))
            .MidSignal(width < 3 ? 0.0f : min(maxSignal, max(0.0f, raw_mid.Get<id>())))
            .MaxSignal(min(maxSignal, max(0.0f, signalMax_[threadIdx.x].template Get<id>())))
            .SignalM2(signalM2_[threadIdx.x].template Get<id>())
            .Label(LabelToAnalog(FullFrameLabel().template Get<id>()));

        return ret;
    }

    __device__ void ResetSegment(PBBool2 boundaryMask, uint32_t frameIndex,
                      PBShort2 label, PBShort2 signal)
    {
        startFrame_[threadIdx.x] = Blend(boundaryMask, frameIndex, startFrame_[threadIdx.x]);
        signalFrstFrame_[threadIdx.x] = Blend(boundaryMask, signal, signalFrstFrame_[threadIdx.x]);
        signalLastFrame_[threadIdx.x] = Blend(boundaryMask, 0, signalLastFrame_[threadIdx.x]);
        signalMax_[threadIdx.x] = Blend(boundaryMask, signal, signalMax_[threadIdx.x]);
        signalTotal_[threadIdx.x] = Blend(boundaryMask, 0, signalTotal_[threadIdx.x]);
        signalM2_[threadIdx.x] = Blend(boundaryMask, 0, signalM2_[threadIdx.x]);
        label_[threadIdx.x] = Blend(boundaryMask, label, label_[threadIdx.x]);
    }

    __device__ void AddSignal(PBBool2 update, PBShort2 signal)
    {
        signalTotal_[threadIdx.x] = Blend(update,
                                          signalTotal_[threadIdx.x] + signalLastFrame_[threadIdx.x],
                                          signalTotal_[threadIdx.x]);
        signalM2_[threadIdx.x] = Blend(update,
                                       signalM2_[threadIdx.x] + pow2(signalLastFrame_[threadIdx.x]),
                                       signalM2_[threadIdx.x]);
        signalLastFrame_[threadIdx.x] = Blend(update, signal, signalLastFrame_[threadIdx.x]);
        signalMax_[threadIdx.x] = Blend(update, max(signal, signalMax_[threadIdx.x]), signalMax_[threadIdx.x]);
    }

private:
    CudaArray<uint2, blockThreads> startFrame_;        // Needed because of partial segments
    CudaArray<uint2, blockThreads> endFrame_;          // 1 + the last frame included in the segment

    CudaArray<PBShort2, blockThreads> signalFrstFrame_;   // Signal of the most recent frame added
    CudaArray<PBShort2, blockThreads> signalLastFrame_;   // Signal recorded for the last frame in the segment
    CudaArray<PBHalf2, blockThreads> signalMax_;         // Max signal over all frames in segment
    CudaArray<PBShort2, blockThreads> signalTotal_;       // Signal total, excluding the first and last frame
    CudaArray<PBHalf2, blockThreads> signalM2_;          // Sum of squared signals, excluding the first and last frame

    CudaArray<PBShort2, blockThreads> label_;             // // Internal label ID corresponding to detection modes
};

template <size_t blockThreads>
__launch_bounds__(32, 32)
__global__ void ProcessLabels(GpuBatchData<const PBShort2> labels,
                              GpuBatchData<const PBShort2> signal,
                              GpuBatchData<const PBShort2> latSignal,
                              uint32_t firstFrameIdx,
                              DeviceView<Segment<blockThreads>> workingSegments,
                              GpuBatchVectors<Data::Pulse> pulsesOut)
{
    assert(labels.NumFrames() == signal.NumFrames() + latSignal.NumFrames());

    // TODO: move this to shared mem?
    auto& segment = workingSegments[blockIdx.x];

    // each thread handles 2 zmw, which normally are interleaved in something like PBShort2,
    // but cannot be for pulses
    auto pulsesZmw1 = pulsesOut.GetVector(blockIdx.x*2*blockDim.x + threadIdx.x*2);
    auto pulsesZmw2 = pulsesOut.GetVector(blockIdx.x*2*blockDim.x + threadIdx.x*2+1);
    pulsesZmw1.Reset();
    pulsesZmw2.Reset();

    auto labelZmw = labels.ZmwData(blockIdx.x, threadIdx.x);
    auto signalZmw = latSignal.ZmwData(blockIdx.x, threadIdx.x);

    auto HandleFrame = [&](PBShort2 label, PBShort2 signal, uint32_t frame) {
        auto boundaryMask = segment.IsNewSegment(label);
        auto pulseMask = segment.IsPulse();

        auto emit = boundaryMask && pulseMask;
        if (emit.X())
        {
            pulsesZmw1.push_back(segment.ToPulse<0>(frame));
        }
        if (emit.Y())
        {
            pulsesZmw2.push_back(segment.ToPulse<1>(frame));
        }

        segment.ResetSegment(boundaryMask, frame, label, signal);
        segment.AddSignal(!boundaryMask, signal);
    };

    const int latFrames = latSignal.NumFrames();
    for (int i = 0; i < latFrames; i++)
    {
        HandleFrame(labelZmw[i], signalZmw[i], i + firstFrameIdx);
    }

    signalZmw = signal.ZmwData(blockIdx.x, threadIdx.x);
    for (int i = 0; i < signal.NumFrames(); i++)
    {
        HandleFrame(labelZmw[i+latFrames], signalZmw[i], i + latFrames + firstFrameIdx);
    }
}

}

class DevicePulseAccumulator::AccumImpl
{
    static constexpr size_t blockThreads = laneSize / 2;
public:
    AccumImpl(size_t lanesPerPool)
        : workingSegments_(lanesPerPool)
    {
    }

    PulseBatch Process(const PulseBatchFactory& factory, LabelsBatch labels)
    {
        static constexpr size_t threadsPerBlock = 32;
        assert(threadsPerBlock*2 == labels.LaneWidth());
        auto ret = factory.NewBatch(labels.Metadata());
        ProcessLabels<threadsPerBlock><<<labels.LanesPerBatch(),threadsPerBlock>>>(
                labels,
                labels.TraceData(),
                labels.LatentTrace(),
                labels.Metadata().FirstFrame(),
                workingSegments_.GetDeviceView(),
                ret.Pulses());

        Cuda::CudaSynchronizeDefaultStream();
        return ret;
    }

private:
    DeviceOnlyArray<Segment<blockThreads>> workingSegments_;
};

void DevicePulseAccumulator::Configure(size_t maxCallsPerZmw)
{
    constexpr bool hostExecution = false;
    PulseAccumulator::InitAllocationPools(hostExecution, maxCallsPerZmw);
}

void DevicePulseAccumulator::Finalize()
{
    PulseAccumulator::DestroyAllocationPools();
}

DevicePulseAccumulator::DevicePulseAccumulator(uint32_t poolId, uint32_t lanesPerPool)
    : PulseAccumulator(poolId)
    , impl_(std::make_unique<AccumImpl>(lanesPerPool))
{

}

DevicePulseAccumulator::~DevicePulseAccumulator() = default;

Data::PulseBatch DevicePulseAccumulator::Process(Data::LabelsBatch labels)
{
    return impl_->Process(*batchFactory_, std::move(labels));
}

}}}     // namespace PacBio::Mongo::Basecaller

